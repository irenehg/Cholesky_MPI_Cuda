#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

using namespace std;
using std::vector;


//Kernel encargado de los elementos de la diagonal
__global__ void Diagonal(float* matriz_a, float* matriz_l, int n, int numCol) {

    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Solo sacamos elementos de la diagonal principal
    if (fila < n && col < n) {
        if (fila == col && col == numCol) {
            float diagSum = 0.0f;
            for (int k = 0; k < col; k++) {
                diagSum += (matriz_l[col * n + k]) * (matriz_l[col * n + k]);
            }
            matriz_l[fila * n + col] = sqrt((float)(matriz_a[fila * n + col] - diagSum));
        }
    }
}

//kernel encargado de los elementos de la columna
__global__ void Columna(float* matriz_a, float* matriz_l, int n, int num) {

    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Elementos de la columna por debajo de la diagonal principal
    if (col == num && fila > col && fila < n) {
        float sum = 0.0f;
        for (int k = 0; k < col; k++) {
            sum += (matriz_l[fila * n + k] * matriz_l[col * n + k]);
        }
        matriz_l[fila * n + col] = (matriz_a[fila * n + col] - sum) / matriz_l[col * n + col];
    }
}



int main(int argc, char *argv[]){
  int n, i, j, k, p, c, r;
  double sum,aux;


  n = 4; //por defecto tam de la matriz

  if(argc != 1){    //si nos pasan parametro, es el tam de la matriz
    n = atoi(argv[1]);
  }

  size_t bytes = n * n * sizeof(int);


  //Partimos de una matrz A simetrica definida positiva (A*x=B)
  vector<float> a(n * n);  //no he usado malloc como en la version de mpi porque me daba error

  //Rellenamos valores de A con floats aleatorios en la triangular inferior
  for(i=0; i<n; i++){
    for(j=0; j<i; j++){
      a[i*n+j]=(rand() %9)+1.0;
    }
  }

  //Completamos la triangular superior de forma que sea simetrica
  for(i=0; i<n; i++){
    for(j=i+1; j<n; j++){
      a[i*n+j]=a[j*n+i];
    }
  }

  //Completamos con elementos de la diagonal de forma que ningun 'subdeterminante' sea <=0
  for (i=0; i< n; i++) {
    double s = 0.0;
    for (j=0; j< i; j++)
      s += a[i*n+j];

    for (j= i+1; j< n; j++)
      s += a[j*n+i];

    a[i*n+i] = s *5;
  }

  // a[0] = 4;
  // a[1] = -1;
  // a[2] = 0;
  // a[3] = 2;
  // a[4] = -1;
  // a[5] = 4;
  // a[6] = -1;
  // a[7] = 0;
  // a[8] = 0;
  // a[9] = -1;
  // a[10] = 4;
  // a[11] = 1;
  // a[12] = 2;
  // a[13] = 0;
  // a[14] = 1;
  // a[15] = 3;



  // cout << "\nMostramos matriz origen:" << endl;
  // printf("\n A \n");
  // for(i = 0; i < n; i++){
  //   for(j=0; j<n;j++)
  //     printf("%f ",a[i*n+j]);
  //   printf ( "\n");
  // }


  //Descomponemos A en matriz triangular inferior L y triangular superior U
  vector<float> l(n * n, 0.0f);
  for(i=0; i < n; i++){
    for(j = 0; j < n; j++){
      if(j<=i)
        l[i*n+j]=a[i*n+j];
      else
        l[i*n+j] = 0.0;
    }
  }


  // Reservamos espacio en la GPU para ambas matrices
	float* d_A, * d_L;
  hipMalloc(&d_A, bytes);
  hipMalloc(&d_L, bytes);


  // Rellenamos las matrices creadas en GPU con valores
  hipMemcpy(d_A, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_L, l.data(), bytes, hipMemcpyHostToDevice);


  int threads = 64;
  int blocks = 240;
  dim3 threadsPerBlock(threads, threads);
  int numBlocks = 1;

  //inicializamos reloj
	struct timespec cgt1,cgt2;
	double ncgt; //para tiempo de ejecución

  clock_gettime(CLOCK_REALTIME,&cgt1);


  for (int i = 0; i < n; i++) {
      // Kernel que calcula elemento de la diagonal
      Diagonal << <numBlocks, threadsPerBlock >> > (d_A, d_L, n, i);
      // Kernel que calcula los elementos de esa columna
      Columna << <numBlocks, threadsPerBlock >> > (d_A, d_L, n, i);
  }

  clock_gettime(CLOCK_REALTIME,&cgt2);
  ncgt=(double) (cgt2.tv_sec-cgt1.tv_sec)+ (double) ((cgt2.tv_nsec-cgt1.tv_nsec)/(1.e+9));

  hipMemcpy(l.data(), d_L, bytes, hipMemcpyDeviceToHost);

  // cout << "Cholesky en GPU:" << endl;
  // cout << "\nMostramos ahora la descomposición:" << endl;
  // printf("\n L \n");
  // for(i = 0; i < n; i++){
  //   for(j=0; j<n;j++)
  //     printf("%f ",l[i*n+j]);
  //   printf ( "\n");
  // }
  cout << "\nTamaño: " << n << "\tTiempo GPU: " << ncgt << "s\n";
  cout << "\nBloques: "<< blocks << "\thebras por bloque: " << threads << "\n";
  cout << endl;

  // Liberamos memoria
  hipFree(d_A);
  hipFree(d_L);

}
